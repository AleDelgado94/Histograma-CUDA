/*******************************
 * Autor: Alejandro Delgado Martel
 * Nombre: Proyecto Final Versión 1
 *******************************/


#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>

#define HISTO_ELEMENTS 1000


__global__ void inicializa_histograma(float* A, float* hist, int num_elements, int hist_elements){
	int i = (blockIdx.x * blockDim.x + threadIdx.x);


	if(i < num_elements){
		if(i==0){
			for(int j=0; j<hist_elements; j++)
				hist[j]=0.0;
		}
	}
}

__global__ void histograma(float* A, float* hist, int num_elements, int hist_elements){


	//Posicion del thread
	int i = (blockIdx.x * blockDim.x + threadIdx.x);


	if(i < num_elements){

		int pos = (int)(fmod(A[i],(float)hist_elements));
		atomicAdd(&(hist[pos]), 1.0);
	}
}



void fError(hipError_t err){
	if(err != hipSuccess){
		printf("Ha ocurrido un error el la linea %d con codigo: %s\n", __LINE__, hipGetErrorString(err));
	}
}


int main(){

	//cudaSetDevice(0);

	int num_elements = 1000000;
	int hist_elements = HISTO_ELEMENTS;

	//Reservar espacio en memoria HOST


	float * h_A = (float*)malloc(num_elements * sizeof(float));

	float * h_hist = (float*)malloc(hist_elements * sizeof(float));


	if(h_A == NULL ){
		printf("Error al reservar memoria para los vectores HOST");
		exit(1);
	}



	//Inicializar elementos de los vectores de forma hormogenea
	for(int i=0; i<num_elements; i++){
		h_A[i] = (float)i;
	}

	//Inicializamos a 0 todas las posiciones del histograma
	//for(int i=0; i<hist_elements; i++)
		//h_hist[i]=0.0;


	hipError_t err;

	int size = num_elements * sizeof(float);
	int size_hist = hist_elements * sizeof(float);


	float * d_A = NULL;
	err = hipMalloc((void **)&d_A, size);
	fError(err);

	float * d_hist = NULL;
	err = hipMalloc((void**)&d_hist, size_hist);
	fError(err);


	//Copiamos a GPU DEVICE
	err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
	err = hipMemcpy(d_hist, h_hist, size_hist, hipMemcpyHostToDevice);


	int HilosPorBloque = 512;
	int BloquesPorGrid = (num_elements + HilosPorBloque -1) / HilosPorBloque;


	hipError_t Err;

	//Lanzamos el kernel y medimos tiempos
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);

	inicializa_histograma<<<BloquesPorGrid, HilosPorBloque>>>(d_A, d_hist, num_elements, hist_elements);
	histograma<<<BloquesPorGrid, HilosPorBloque>>>(d_A,d_hist, num_elements, hist_elements);


	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	float tiempo_reserva_host;
	hipEventElapsedTime(&tiempo_reserva_host, start, stop);

	Err = hipGetLastError();
	fError(Err);


	printf("Tiempo de suma vectores DEVICE: %f\n", tiempo_reserva_host);

	hipEventDestroy(start);
	hipEventDestroy(stop);


	//Copiamos a CPU el vector C
	err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
	fError(err);

	err = hipMemcpy(h_hist, d_hist, size_hist, hipMemcpyDeviceToHost);
	fError(err);



	float suma = 0;

	for(int i=0; i<hist_elements; i++){
		//printf("%f \n", h_hist[i]);
		//printf("\n");
		suma = suma + h_hist[i];
	}

	printf("La suma es: %f", suma);

}

